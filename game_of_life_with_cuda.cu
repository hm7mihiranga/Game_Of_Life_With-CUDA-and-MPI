#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>
#include <SDL2/SDL.h>

#define BLOCK_SIDE 16
#define WINDOW_WIDTH 640
#define WINDOW_HEIGHT 640

typedef unsigned char ubyte;

// CUDA device functions
__device__ ubyte getat(ubyte* pboard, int nrows, int ncols, int x, int y) {
    if (x >= 0 && x < ncols && y >= 0 && y < nrows)
        return pboard[x * nrows + y];
    return 0x0;
}

__device__ int numneighbors(int x, int y, ubyte* pboard, int nrows, int ncols) {
    int num = 0;
    num += (getat(pboard, nrows, ncols, x-1, y));
    num += (getat(pboard, nrows, ncols, x+1, y));
    num += (getat(pboard, nrows, ncols, x, y-1));
    num += (getat(pboard, nrows, ncols, x, y+1));
    num += (getat(pboard, nrows, ncols, x-1, y-1));
    num += (getat(pboard, nrows, ncols, x-1, y+1));
    num += (getat(pboard, nrows, ncols, x+1, y-1));
    num += (getat(pboard, nrows, ncols, x+1, y+1));
    
    return num;
}

__global__ void simstep(int nrows, int ncols, ubyte* pCurrBoard, ubyte* pNewBoard) {
    int x = blockIdx.x * BLOCK_SIDE + threadIdx.x;
    int y = blockIdx.y * BLOCK_SIDE + threadIdx.y;

    if (x < ncols && y < nrows) {
        int indx = x * nrows + y;
        pNewBoard[indx] = pCurrBoard[indx];

        int neighbors = numneighbors(x, y, pCurrBoard, nrows, ncols);

        // Apply game rules:
        if (neighbors < 2)
            pNewBoard[indx] = 0x0;
        else if (neighbors > 3)
            pNewBoard[indx] = 0x0;
        else if (neighbors == 3 && !pCurrBoard[indx])
            pNewBoard[indx] = 0x1;
    }
}

// Function to initialize the random board state
void randomizeBoard(ubyte* pboard, int nrows, int ncols, float probability) {
    for (int x = 0; x < ncols; x++) {
        for (int y = 0; y < nrows; y++) {
            float rnd = rand() / (float)RAND_MAX;
            pboard[x * nrows + y] = (rnd >= probability) ? 0x1 : 0x0;
        }
    }
}

// Function to draw the board using SDL
void drawBoard(SDL_Renderer* renderer, ubyte* pboard, int nrows, int ncols) {
    int cellSize = WINDOW_WIDTH / ncols;
    for (int x = 0; x < ncols; x++) {
        for (int y = 0; y < nrows; y++) {
            SDL_Rect cellRect = { x * cellSize, y * cellSize, cellSize, cellSize };
            SDL_SetRenderDrawColor(renderer, pboard[x * nrows + y] * 255, pboard[x * nrows + y] * 255, 0, 255);
            SDL_RenderFillRect(renderer, &cellRect);
        }
    }
}

// Function to toggle the state of a cell
void toggleCell(ubyte* pboard, int nrows, int x, int y) {
    int index = x * nrows + y;
    pboard[index] = (pboard[index] == 0x1) ? 0x0 : 0x1;
}

// Main simulation function to be called from MPI code
extern "C" void run_simulation(int boardW, int boardH, int ngenerations) {
    srand(time(0));
    ubyte* pboard = (ubyte*)malloc(boardW * boardH * sizeof(ubyte));
    randomizeBoard(pboard, boardH, boardW, 0.7f);

    SDL_Init(SDL_INIT_VIDEO);
    SDL_Window* window = SDL_CreateWindow("Game of Life", SDL_WINDOWPOS_UNDEFINED, SDL_WINDOWPOS_UNDEFINED, WINDOW_WIDTH, WINDOW_HEIGHT, SDL_WINDOW_SHOWN);
    SDL_Renderer* renderer = SDL_CreateRenderer(window, -1, SDL_RENDERER_ACCELERATED);

    ubyte* pDevBoard0;
    hipMalloc((void**)&pDevBoard0, boardW * boardH * sizeof(ubyte));
    hipMemcpy(pDevBoard0, pboard, boardH * boardW * sizeof(ubyte), hipMemcpyHostToDevice);

    ubyte* pDevBoard1;
    hipMalloc((void**)&pDevBoard1, boardW * boardH * sizeof(ubyte));
    hipMemset(pDevBoard1, 0x0, boardH * boardW * sizeof(ubyte));

    dim3 blocksize(BLOCK_SIDE, BLOCK_SIDE);
    dim3 gridsize((boardW + BLOCK_SIDE - 1) / BLOCK_SIDE, (boardH + BLOCK_SIDE - 1) / BLOCK_SIDE);

    struct timeval ti;
    gettimeofday(&ti, NULL);

    ubyte* pcurr;
    ubyte* pnext;
    bool running = true;
    SDL_Event event;
    bool quit = false;

    int gen = 0;

    while (!quit) {
        while (SDL_PollEvent(&event)) {
            if (event.type == SDL_QUIT) {
                quit = true;
            }
        }

        if (gen < ngenerations) {
            pcurr = (gen % 2 == 0) ? pDevBoard0 : pDevBoard1;
            pnext = (gen % 2 == 0) ? pDevBoard1 : pDevBoard0;

            simstep<<<gridsize, blocksize>>>(boardH, boardW, pcurr, pnext);
            hipDeviceSynchronize();

            hipMemcpy(pboard, pnext, boardH * boardW * sizeof(ubyte), hipMemcpyDeviceToHost);
            SDL_RenderClear(renderer);
            drawBoard(renderer, pboard, boardH, boardW);
            SDL_RenderPresent(renderer);

            gen++;
        } else {
            quit = true;
        }
    }

    hipMemcpy(pboard, pcurr, boardW * boardH * sizeof(ubyte), hipMemcpyDeviceToHost);

    SDL_DestroyRenderer(renderer);
    SDL_DestroyWindow(window);
    SDL_Quit();

    hipFree(pDevBoard0);
    hipFree(pDevBoard1);
    free(pboard);

    struct timeval tf;
    gettimeofday(&tf, NULL);
    double t = ((tf.tv_sec - ti.tv_sec) * 1000.0) + ((tf.tv_usec - ti.tv_usec) / 1000.0);
    printf("Simulation completed in %f ms\n", t);
}
